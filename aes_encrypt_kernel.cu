#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

static uint16_t aes_field = 283;
__device__ static uint16_t aes_field_gpu = 283;
static uint8_t* sbox;
// __device__ __constant__ static uint8_t* sbox_gpu;
// static uint8_t inv_sbox[0x100];
__device__ static uint8_t mix_columns_matrix[4][4] = {{2, 1, 1, 3}, {3, 2, 1, 1}, {1, 3, 2, 1}, {1, 1, 3, 2}};
// static uint8_t mix_columns_matrix[4][4] = {{2, 1, 1, 3}, {3, 2, 1, 1}, {1, 3, 2, 1}, {1, 1, 3, 2}};
// static uint8_t inv_mix_columns[4][4] = {{14, 9, 13, 11}, {11, 14, 9, 13}, {13, 11, 14, 9}, {9, 13, 11, 14}};
uint8_t round_constant[4] = {1, 0, 0, 0};


struct aes_encoder {
    uint8_t key_size;
    uint8_t* expanded_key;
};

inline hipError_t checkCuda(hipError_t result, int line)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error at line %d: %s\n", line, hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

void rotate_left(uint8_t* bytes) {
    uint8_t first_byte = bytes[1];
    bytes[1] = bytes[2];
    bytes[2] = bytes[3];
    bytes[3] = bytes[0];
    bytes[0] = first_byte;
}

__device__ void sub_bytes_gpu(uint8_t* bytes, int len_bytes, uint8_t* sbox_gpu) {
    for (int i = 0; i < len_bytes; i++) {
        bytes[i] = sbox_gpu[bytes[i]];
    }
}

void sub_bytes(uint8_t* bytes, int len_bytes) {
    for (int i = 0; i < len_bytes; i++) {
        bytes[i] = sbox[bytes[i]];
    }
}

void add_round_constant(uint8_t* bytes) {
    bytes[0] ^= round_constant[0];
    bytes[1] ^= round_constant[1];
    bytes[2] ^= round_constant[2];
    bytes[3] ^= round_constant[3];
    uint16_t new_constant = (uint16_t)round_constant[0] << 1;
    if (new_constant >= 0x100) {
        new_constant ^= aes_field;
    }
    round_constant[0] = (uint8_t)new_constant;
}

// Currently assuming small key.
void key_expansion(struct aes_encoder* aes, uint8_t* initial_key, int key_size) {
    checkCuda(hipMallocManaged(&(aes->expanded_key), 176 * sizeof(uint8_t)), 66);
    // checkCuda(hipMemcpy(aes->expanded_key, initial_key, 4 * sizeof(int), hipMemcpyHostToDevice), 67);
    for (int i = 0; i < key_size; i++) {
        // printf("Copy byte: %d\n", i);
        aes->expanded_key[i] = initial_key[i];
    }
    int round = 0;
    while (round < 10) {
        // printf("Round: %d\n", round);
        int current_key_size = 16 * (round + 1);
        for (int k = 0; k < 4; k++) {
            int i = current_key_size - 4 + (k * 4);
            uint8_t last_four_bytes[] = {aes->expanded_key[i], aes->expanded_key[i + 1], aes->expanded_key[i + 2], aes->expanded_key[i + 3]};
            if (k == 0) {
                rotate_left(last_four_bytes);
                sub_bytes(last_four_bytes, 4);
                add_round_constant(last_four_bytes);
            }
            i = current_key_size - 16 + (k * 4);
            uint8_t first_four_of_last_sixteen_bytes[] = {aes->expanded_key[i], aes->expanded_key[i + 1], aes->expanded_key[i + 2], aes->expanded_key[i + 3]};
            aes->expanded_key[current_key_size + (4 * k)] = first_four_of_last_sixteen_bytes[0] ^ last_four_bytes[0];
            aes->expanded_key[current_key_size + (4 * k) + 1] = first_four_of_last_sixteen_bytes[1] ^ last_four_bytes[1];
            aes->expanded_key[current_key_size + (4 * k) + 2] = first_four_of_last_sixteen_bytes[2] ^ last_four_bytes[2];
            aes->expanded_key[current_key_size + (4 * k) + 3] = first_four_of_last_sixteen_bytes[3] ^ last_four_bytes[3];
        }
        round++;
    }
}

struct aes_encoder* aes_init(int key_size, int* initial_key) {
    struct aes_encoder* e; 
    checkCuda(hipMallocManaged(&e, sizeof(struct aes_encoder)), 92);
    return e;
}

__device__ void shift_rows(uint8_t* bytes) {
    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < i; j++) {
            uint8_t temp = bytes[0 + i];
            bytes[0 + i] = bytes[4 + i];
            bytes[4 + i] = bytes[8 + i];
            bytes[8 + i] = bytes[12 + i];
            bytes[12 + i] = temp;
        }
    }
}

__device__ uint8_t product(uint16_t a, uint16_t b) {
    uint8_t ret = 0;
    for (int i = 0; i < 8; i++) {
        if ((b & 1) == 1) {
            ret ^= a;
        }
        if ((a & 0x80) == 0x80) {
            a <<= 1;
            a ^= aes_field_gpu;
        }
        else {
            a <<= 1;
        }
        b >>= 1;
    }
    return ret;
}

__device__ void mix_columns(uint8_t* bytes) {
    uint8_t* storage = (uint8_t*)malloc(16 * sizeof(uint8_t));
    // hipMallocManaged(&storage, 16 * sizeof(uint8_t));
    for (int i = 0; i < 16; i++) {
        storage[i] = 0;
    }
    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            storage[j + (4 * i)] = product(mix_columns_matrix[0][j], bytes[(i * 4)]) ^ 
                product(mix_columns_matrix[1][j], bytes[(i * 4) + 1]) ^ 
                product(mix_columns_matrix[2][j], bytes[(i * 4) + 2]) ^
                product(mix_columns_matrix[3][j], bytes[(i * 4) + 3]);
        }
    }
    for (int i = 0; i < 16; i++) {
        bytes[i] = storage[i];
    }
}

__device__ void add_round_key(uint8_t* bytes, uint8_t* key_section) {
    for (int i = 0; i < 16; i++) {
        // printf("%d\n", threadIdx.x);
        bytes[i] ^= key_section[i];
    }
}

__device__ void encrypt_16(uint8_t* plain_text, struct aes_encoder* aes, uint8_t* sbox) {
        uint8_t* cipher_text = plain_text;

        int round = 0;
        while (round < 10) {
            uint8_t* key_section = aes->expanded_key + (round * 16); // I love pointer arithmetic!
            add_round_key(cipher_text, key_section);
            sub_bytes_gpu(cipher_text, 16, sbox);
            shift_rows(cipher_text);
            if (round != 9) {
                mix_columns(cipher_text);
            }
            else {
                key_section = aes->expanded_key + 160;
                add_round_key(cipher_text, key_section);
            }
            round++;
        }
}

__global__ void encrypt(uint8_t* plain_text, struct aes_encoder* aes, int length, uint8_t* sbox) {
    for (int i = blockIdx.x * 16; i < length; i += (gridDim.x * 16)) {
        encrypt_16(plain_text + i, aes, sbox);
    }
}

void sbox_init() {
    checkCuda(hipMallocManaged(&sbox, 0x100), 189);
    FILE* sbox_file = fopen("sbox.txt", "r");
    char s[6];
    while ((fgets(s, 7, sbox_file))) {
        char i[3] = {s[0], s[1], 0};
        char sub[3] = {s[3], s[4], 0};
        int i_int = (int)strtol(i, NULL, 16);
        int sub_int = (int)strtol(sub, NULL, 16);
        sbox[i_int] = sub_int;
    }
    fclose(sbox_file);
}

int main(int ac, char** av) {
    sbox_init();
    // checkCuda(hipMallocManaged(&sbox_gpu, 0x100), 190);
    // checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(sbox_gpu), sbox, 0x100), 201);
    // hipDeviceSynchronize();
    struct aes_encoder* aes = aes_init(0, 0);
    const TCHAR* key_file = TEXT("aes_key");
    HANDLE h_file = CreateFile(key_file, GENERIC_READ, 0, NULL, OPEN_EXISTING, FILE_ATTRIBUTE_NORMAL, 0);
    HANDLE h_map = CreateFileMapping(h_file, NULL, PAGE_READONLY, 0, 0, NULL);
    const TCHAR* pt_file = TEXT("plain_text_long");
    HANDLE pt_h_file = CreateFile(pt_file, GENERIC_ALL, 0, NULL, OPEN_EXISTING, FILE_ATTRIBUTE_NORMAL, 0);
    if (pt_h_file == INVALID_HANDLE_VALUE) {
        printf("Problem opening file \n");
    }
    HANDLE pt_h_map = CreateFileMapping(pt_h_file, NULL, PAGE_READWRITE, 0, 0, NULL);
    if (pt_h_map == NULL) {
        printf("Problem creating mapping of file \n");
    }
    uint8_t* initial_key = (uint8_t*)MapViewOfFile(h_map, FILE_MAP_READ, 0, 0, 0);
    hipMallocManaged(&initial_key, 16 * sizeof(uint8_t));

    for (int i = 0; i < 16; i++) {
        initial_key[i] = i + (0x10 * i);
        printf("%02X", initial_key[i]);
    }
    printf("\n");
    uint8_t* base_pt = (uint8_t*)MapViewOfFile(pt_h_map, FILE_MAP_READ | FILE_MAP_WRITE, 0, 0, 0);
    uint8_t* test;
    int length = 16000;
    checkCuda(hipMallocManaged(&test, length), 211);
    checkCuda(hipMemcpy(test, base_pt, length, hipMemcpyHostToDevice), 212);
    if (test == NULL) {
        printf("Problem mapping view of file \n");
    }
    // printf("Reached key expansion\n");
    key_expansion(aes, initial_key, 16);
    int device_id;
    int number_of_SMs;
    hipGetDevice(&device_id);
    hipDeviceGetAttribute(&number_of_SMs, hipDeviceAttributeMultiprocessorCount, device_id);
    // How many threads do we need?
    // We've initialized the length to be 16000 (not the best practice, but just for the sake of this project to measure performance boosts.)
    // Each plaintext block is 16 bytes. So 1000 threads should be able to parallelize perfectly fine.
    // To be safe, I'll implement a stepping as well for each thread.
    // printf("Reached encrypt\n");
    int num_blocks = 32 * number_of_SMs;
    encrypt<<<num_blocks, 1>>>(test, aes, length, sbox);
    // printf("Passed encrypt\n");
    checkCuda(hipDeviceSynchronize(), 227);
    // printf("Checking first error\n");
    checkCuda(hipGetLastError(), 226);
    // printf("Printing block\n");
    for (int i = 0; i < 16; i++) {
        printf("%02X", test[i]);
    }
    // printf("Closing handles\n");
    CloseHandle(h_map);
    CloseHandle(h_file);
    CloseHandle(pt_h_map);
    CloseHandle(pt_h_file);
    return 0;
}